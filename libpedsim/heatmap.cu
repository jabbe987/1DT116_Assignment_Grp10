#include "hip/hip_runtime.h"
// Created for Low Level Parallel Programming 2017
//
// Implements the heatmap functionality. 
//

#include "ped_model.h"
#include "ped_agent.h"
#include "hip/hip_runtime.h"
#include "ped_agents.h"
#include ""

#include <cstdlib>
#include <iostream>
#include <cmath>
using namespace std;

// Memory leak check with msvc++
#include <stdlib.h>

// #define SIZE 256
// #define SCALED_SIZE (SIZE * 2)

// CUDA kernel for scaling the heatmap.
// The input 'd_heatmap' is a 1D array representing the original heatmap of size SIZE x SIZE.
// The output 'd_scaled' is a 1D array for the scaled heatmap of size SCALED_SIZE x SCALED_SIZE,
// where SCALED_SIZE = SIZE * CELLSIZE.

// static hipStream_t updateStream = nullptr;


inline void safe_call(hipError_t err) {
    if (err != hipSuccess) {
        cerr << "CUDA error: " << hipGetErrorString(err) << endl;
    }
}

__global__ void fadeKernel(int *d_hm, int length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        d_hm[idx] = (int)round(d_hm[idx] * 0.80); //TODO rounding
        // d_heatmap[idx] = (int)(d_heatmap[idx]*0.80f+0.5f); // 0.5f is for rounding. 8.1+0.5=8.6 -> 8, but 8.6+0.5=9
    }
}

__global__ void addAgentHeatKernel(int* d_hm, int size,
    const int* d_agentDesiredX,
    const int* d_agentDesiredY,
    int numAgents) {

     // printf("CUDA_AgentHeat ");
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numAgents) {
        int x = d_agentDesiredX[idx];
        int y = d_agentDesiredY[idx];
        // printf("Agent %d at (%d, %d)\n", idx, x, y);
        if (x >= 0 && x < size && y >= 0 && y < size) {
            atomicAdd(&d_hm[y*size+x], 40); //TODO checka atomicAdd om korrekt
            // printf("Heatmap[%d][%d] = %d\n", x, y, d_heatmap[y * size + x]);
        }
    }
}

__global__ void limitHeatmapValueKernel(int* d_hm, int length) {
    // printf("CUDA_Limit ");
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length && d_hm[idx] > 255) {
        d_hm[idx]=255;
    }
}


__global__ void scaleHeatmapKernel(const int *d_hm, int *d_shm, int size, int cellsize) {
    int scaled_size = size * cellsize;
    int scaled_index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = scaled_size * scaled_size;
    
    if (scaled_index < total_elements) {
        // Determine the (x, y) coordinate in the scaled heatmap.
        int scaled_y = scaled_index / scaled_size;
        int scaled_x = scaled_index % scaled_size;
        
        // Map back to the original heatmap coordinate.
        int orig_x = scaled_x / cellsize;
        int orig_y = scaled_y / cellsize;
        int orig_index = orig_y * size + orig_x;
        
        // Each pixel in the scaled image gets the value from the corresponding original cell.
        d_shm[scaled_index] = d_hm[orig_index];
    }
}

// 1D blur kernel: No shared memory tiles, each thread handles one pixel
__global__ void blurHeatmapKernel(const int* d_in, int* d_out, int width, const int* d_weights)
{
    // Compute the global 1D index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we don't read/write out of bounds
    int total = width * width;
    if (idx >= total) return;

    // Convert 1D index -> (x, y)
    int x = idx % width;
    int y = idx / width;

    // Only blur if we can safely access a 5×5 region around (x, y)
    if (x >= 2 && x < width - 2 && y >= 2 && y < width - 2) {
        int sum = 0;
        // Accumulate weighted sum from 5×5 neighborhood
        for (int dy = -2; dy <= 2; dy++) {
            for (int dx = -2; dx <= 2; dx++) {
                int neighborVal = d_in[(y + dy) * width + (x + dx)];
                int weight      = d_weights[(dy + 2) * 5 + (dx + 2)];
                sum += neighborVal * weight;
            }
        }
        // Store ARGB result (red channel, alpha == sum/273)
        d_out[idx] = 0x00FF0000 | ((sum / 273) << 24);
    }
}

__global__ void initializeHeatmap(int *d_hm, int *d_shm, int *d_bhm, int length, int scaled_length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        d_hm[idx] = 0;
    }
    if (idx < scaled_length) {
        d_shm[idx] = 0;
        d_bhm[idx] = 0;
    }
}

void Ped::Model::setupHeatmap() {
    int *d_hm, *d_shm, *d_bhm;
    hipStream_t stream;
    hipStreamCreate(&stream);

    int length = SIZE * SIZE;
    int scaled_length = SCALED_SIZE * SCALED_SIZE;

    // Allocate GPU memory
    safe_call(hipMalloc((void**)&d_hm, SIZE * SIZE * sizeof(int)));
    safe_call(hipMalloc((void**)&d_shm, SCALED_SIZE * SCALED_SIZE * sizeof(int)));
    safe_call(hipMalloc((void**)&d_bhm, SCALED_SIZE * SCALED_SIZE * sizeof(int)));

    // Define thread block and grid sizes
    int threadsPerBlock = THREADSPERBLOCK;
    int totalElements = max(length, scaled_length);
    int numBlocks = (totalElements + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the CUDA kernel
    initializeHeatmap<<<numBlocks, threadsPerBlock>>>(d_hm, d_shm, d_bhm, length, scaled_length);
    safe_call(hipDeviceSynchronize());
    // Allocate host memory and set up pointers
    int *hm = (int*)malloc(SIZE * SIZE * sizeof(int));
    int *shm = (int*)malloc(SCALED_SIZE * SCALED_SIZE * sizeof(int));
    int *bhm = (int*)malloc(SCALED_SIZE * SCALED_SIZE * sizeof(int));

    heatmap = (int**)malloc(SIZE * sizeof(int*));
    scaled_heatmap = (int**)malloc(SCALED_SIZE * sizeof(int*));
    blurred_heatmap = (int**)malloc(SCALED_SIZE * sizeof(int*));

    // Copy initialized data from GPU to CPU
    safe_call(hipMemcpy(hm, d_hm, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost));
    safe_call(hipMemcpy(shm, d_shm, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost));
    safe_call(hipMemcpy(bhm, d_bhm, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost));

    // Set up 2D pointers
    for (int i = 0; i < SIZE; i++) {
        heatmap[i] = hm + SIZE * i;
    }
    for (int i = 0; i < SCALED_SIZE; i++) {
        scaled_heatmap[i] = shm + SCALED_SIZE * i;
        blurred_heatmap[i] = bhm + SCALED_SIZE * i;
    }

    // Free GPU memory
    safe_call(hipFree(d_hm));
    safe_call(hipFree(d_shm));
    safe_call(hipFree(d_bhm));
    hipStreamDestroy(stream);
}


void Ped::Model::updateHeatmap() {
    hipStream_t stream;
    hipStreamCreate(&stream);
    int length = SIZE * SIZE;
    int scaled_length = SCALED_SIZE * SCALED_SIZE;
    int numAgents = agents->x.size();

    size_t hmSize = length * sizeof(int);
    size_t shmSize = scaled_length * sizeof(int);
    size_t agentSize = numAgents * sizeof(int);

    // Device pointers
    int *d_hm, *d_shm, *d_bhm, *d_agentDesiredX, *d_agentDesiredY;

    // Allocate memory for integer positions
    std::vector<int> agentDesiredX(agents->desiredX.size());
    std::vector<int> agentDesiredY(agents->desiredY.size());

    // Convert float to int before copying to the GPU
    for (size_t i = 0; i < agents->desiredX.size(); i++) {
        agentDesiredX[i] = static_cast<int>(roundf(agents->desiredX[i])); // Use roundf() for better accuracy
        agentDesiredY[i] = static_cast<int>(roundf(agents->desiredY[i]));
    }

    // float *agentDesiredX = agents->desiredX.data();
    // float *agentDesiredY = agents->desiredY.data();

    int *hm = (int*)malloc(hmSize);
    for (int i = 0; i < SIZE; i++) {
        memcpy(hm + i * SIZE, heatmap[i], SIZE * sizeof(int));
    }


    int *shm = (int*)malloc(shmSize);
    int *bhm = (int*)malloc(shmSize);


    hipMallocAsync((void**)&d_hm, hmSize, stream);
    hipMallocAsync((void**)&d_shm, shmSize, stream);
    hipMallocAsync((void**)&d_bhm, shmSize, stream);
    hipMallocAsync((void**)&d_agentDesiredX, agentSize, stream);
    hipMallocAsync((void**)&d_agentDesiredY, agentSize, stream);
    // hipStreamSynchronize(stream);



    hipMemcpyAsync(d_hm, hm, hmSize, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_agentDesiredX, agentDesiredX.data(), numAgents * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_agentDesiredY, agentDesiredY.data(), numAgents * sizeof(int), hipMemcpyHostToDevice, stream);
    // hipStreamSynchronize(stream);

    int threadsPerBlock = THREADSPERBLOCK; // divisible by 32 (warp size)
    int blocksForFade = (length+threadsPerBlock-1) / threadsPerBlock;
    int blocksForAgents = (numAgents+threadsPerBlock-1) / threadsPerBlock;
    int blocks = (scaled_length+threadsPerBlock-1) / threadsPerBlock;
    // dim3 blockDim2D(32, 32); 
    // dim3 gridDim2D((SCALED_SIZE+blockDim2D.x -1) / blockDim2D.x,(SCALED_SIZE+blockDim2D.y -1) / blockDim2D.y);
    // determine the number of grids by SCALED_SIZE/blockDim2D.x and SCALED_SIZE/blockDim2D.y
    // (SCALED_SIZE + blockDim2D.x - 1) / blockDim2D.x to allow for partial blocks
    // size_t sharedMemSize = (blockDim2D.x + 4) * (blockDim2D.y + 4) * sizeof(int); // +4 for halo, 2 on each side

    fadeKernel<<<blocksForFade, threadsPerBlock, 0, stream>>>(d_hm, length);

    addAgentHeatKernel<<<blocksForAgents, threadsPerBlock, 0, stream>>>(d_hm, SIZE, d_agentDesiredX, d_agentDesiredY, numAgents);

    limitHeatmapValueKernel<<<blocksForFade, threadsPerBlock, 0, stream>>>(d_hm, length);

    scaleHeatmapKernel<<<blocks, threadsPerBlock, 0, stream>>>(d_hm, d_shm, SIZE, CELLSIZE);

    int *d_weights;

    int weights[5][5] = {
        {1, 4, 7, 4, 1},
        {4, 16, 26, 16, 4},
        {7, 26, 41, 26, 7},
        {4, 16, 26, 16, 4},
        {1, 4, 7, 4, 1}
    };

    // Flatten the weights
    int h_weights[25];
    for (int i = 0; i < 5; i++)
        for (int j = 0; j < 5; j++)
            h_weights[i * 5 + j] = weights[i][j];

    safe_call(hipMallocAsync(&d_weights, 25 * sizeof(int), stream));
    hipMemcpyAsync(d_weights, h_weights, 25 * sizeof(int), hipMemcpyHostToDevice, stream);

    blurHeatmapKernel<<<blocks, threadsPerBlock, 0, stream>>>(d_shm, d_bhm, SCALED_SIZE, d_weights);

    hipMemcpyAsync(heatmap[0], d_hm, hmSize, hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync(blurred_heatmap[0], d_bhm, shmSize, hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync(scaled_heatmap[0], d_shm, shmSize, hipMemcpyDeviceToHost, stream);
    // hipStreamSynchronize(stream); // CPU waits for GPU to finish before CPU moves on to the next step.
    // Free device memory.
    // printf("---------Async kernel execution complete-----------------\n");
    hipFreeAsync(d_hm, stream);
    hipFreeAsync(d_shm, stream);
    hipFreeAsync(d_bhm, stream);
    hipFreeAsync(d_agentDesiredX, stream);
    hipFreeAsync(d_agentDesiredY, stream);
    hipFreeAsync(d_weights, stream);
    // hipStreamDestroy(stream);
    free(hm);
    free(shm);
    free(bhm);
}

// void Ped::Model::createStream() {
//     hipStreamCreate(&updatestream);
// }

// void Ped::Model::syncHeatmap() {
//     hipStreamSynchronize(updateStream); // CPU waits for GPU to finish before CPU moves on to the next step.
// }

// void Ped::Model::destroyStream() {
//     hipStreamDestroy(updateStream);
// }