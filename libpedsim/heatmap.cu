#include "hip/hip_runtime.h"
// Created for Low Level Parallel Programming 2017
//
// Implements the heatmap functionality. 
//

#include "ped_model.h"
#include "ped_agent.h"
#include "hip/hip_runtime.h"
#include "ped_agents.h"
#include ""

#include <cstdlib>
#include <iostream>
#include <cmath>
using namespace std;

// Memory leak check with msvc++
#include <stdlib.h>

// #define SIZE 256
// #define SCALED_SIZE (SIZE * 2)

// CUDA kernel for scaling the heatmap.
// The input 'd_heatmap' is a 1D array representing the original heatmap of size SIZE x SIZE.
// The output 'd_scaled' is a 1D array for the scaled heatmap of size SCALED_SIZE x SCALED_SIZE,
// where SCALED_SIZE = SIZE * CELLSIZE.

inline void safe_call(hipError_t err) {
    if (err != hipSuccess) {
        cerr << "CUDA error: " << hipGetErrorString(err) << endl
}

// CUDA kernel for Gaussian blur filter using shared memory
__global__ void blurHeatmapKernel(const int *d_scaled_heatmap, int *d_blurred_heatmap, int width, const int *d_weights) {
    __shared__ int tile[32][32];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int tx = threadIdx.x + 2;
    int ty = threadIdx.y + 2;

    if (x < width && y < width) {
        tile[ty][tx] = d_scaled_heatmap[y * width + x];

        if (threadIdx.x < 2 && x >= 2)
            tile[ty][tx - 2] = d_scaled_heatmap[y * width + x - 2];
        if (threadIdx.x >= blockDim.x - 2 && x + 2 < width)
            tile[ty][tx + 2] = d_scaled_heatmap[y * width + x + 2];
        if (threadIdx.y < 2 && y >= 2)
            tile[ty - 2][tx] = d_scaled_heatmap[(y - 2) * width + x];
        if (threadIdx.y >= blockDim.y - 2 && y + 2 < width)
            tile[ty + 2][tx] = d_scaled_heatmap[(y + 2) * width + x];

        __syncthreads();

        if (x >= 2 && y >= 2 && x < width - 2 && y < width - 2) {
            int sum = 0;
            for (int i = -2; i <= 2; i++) {
                for (int j = -2; j <= 2; j++) {
                    sum += d_weights[(i + 2) * 5 + (j + 2)] * tile[ty + i][tx + j];
                }
            }
            d_blurred_heatmap[y * width + x] = 0x00FF0000 | ((sum / 273) << 24);
        }
    }
}




__global__ void scaleHeatmapKernel(const int *d_heatmap, int *d_scaled, int size, int cellsize) {
    int scaled_size = size * cellsize;
    int scaled_index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = scaled_size * scaled_size;
    
    if (scaled_index < total_elements) {
        // Determine the (x, y) coordinate in the scaled heatmap.
        int scaled_y = scaled_index / scaled_size;
        int scaled_x = scaled_index % scaled_size;
        
        // Map back to the original heatmap coordinate.
        int orig_x = scaled_x / cellsize;
        int orig_y = scaled_y / cellsize;
        int orig_index = orig_y * size + orig_x;
        
        // Each pixel in the scaled image gets the value from the corresponding original cell.
        d_scaled[scaled_index] = d_heatmap[orig_index];
    }
}


__global__ void initializeHeatmap(int *hm, int *shm, int *bhm, int length, int scaled_length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        hm[idx] = 0;
    }
    if (idx < scaled_length) {
        shm[idx] = 0;
        bhm[idx] = 0;
    }
}

void Ped::Model::setupHeatmap() {
    int *d_hm, *d_shm, *d_bhm;

    int length = SIZE * SIZE;
    int scaled_length = SCALED_SIZE * SCALED_SIZE;

    // Allocate GPU memory
    safe_call(hipMalloc((void**)&d_hm, SIZE * SIZE * sizeof(int)));
    safe_call(hipMalloc((void**)&d_shm, SCALED_SIZE * SCALED_SIZE * sizeof(int)));
    safe_call(hipMalloc((void**)&d_bhm, SCALED_SIZE * SCALED_SIZE * sizeof(int)));

    // Define thread block and grid sizes
    int threadsPerBlock = THREADSPERBLOCK;
    int totalElements = max(length, scaled_length);
    int numBlocks = (totalElements + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the CUDA kernel
    initializeHeatmap<<<numBlocks, threadsPerBlock>>>(d_hm, d_shm, d_bhm, length, scaled_length);
    safe_call(hipDeviceSynchronize());
    // Allocate host memory and set up pointers
    int *hm = (int*)malloc(SIZE * SIZE * sizeof(int));
    int *shm = (int*)malloc(SCALED_SIZE * SCALED_SIZE * sizeof(int));
    int *bhm = (int*)malloc(SCALED_SIZE * SCALED_SIZE * sizeof(int));

    heatmap = (int**)malloc(SIZE * sizeof(int*));
    scaled_heatmap = (int**)malloc(SCALED_SIZE * sizeof(int*));
    blurred_heatmap = (int**)malloc(SCALED_SIZE * sizeof(int*));

    // Copy initialized data from GPU to CPU
    safe_call(hipMemcpy(hm, d_hm, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost));
    safe_call(hipMemcpy(shm, d_shm, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost));
    safe_call(hipMemcpy(bhm, d_bhm, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost));

    // Set up 2D pointers
    for (int i = 0; i < SIZE; i++) {
        heatmap[i] = hm + SIZE * i;
    }
    for (int i = 0; i < SCALED_SIZE; i++) {
        scaled_heatmap[i] = shm + SCALED_SIZE * i;
        blurred_heatmap[i] = bhm + SCALED_SIZE * i;
    }

    // Free GPU memory
    safe_call(hipFree(d_hm));
    safe_call(hipFree(d_shm));
    safe_call(hipFree(d_bhm));
}

void Ped::Model::scaleHeatmapCUDA() {
    // Assume SIZE and CELLSIZE are defined, and SCALED_SIZE = SIZE * CELLSIZE.
    int numOrig = SIZE * SIZE;
    int numScaled = SCALED_SIZE * SCALED_SIZE;

    // Flatten the 2D heatmap into a contiguous array.
    int *hm_flat = (int*)malloc(numOrig * sizeof(int));
    for (int i = 0; i < SIZE; i++) {
        memcpy(hm_flat + i * SIZE, heatmap[i], SIZE * sizeof(int));
    }

    // Allocate GPU memory.
    int *d_heatmap, *d_scaled;
    safe_call(hipMalloc((void**)&d_heatmap, numOrig * sizeof(int)));
    safe_call(hipMalloc((void**)&d_scaled, numScaled * sizeof(int)));

    // Copy the original heatmap to device.
    // hipMemcpy(d_heatmap, hm_flat, numOrig * sizeof(int), hipMemcpyHostToDevice);
    safe_call(hipMemcpy(d_heatmap, hm_flat, numOrig * sizeof(int), hipMemcpyHostToDevice));


    // Launch the scaling kernel.
    int threadsPerBlock = THREADSPERBLOCK;
    int blocks = (numScaled + threadsPerBlock - 1) / threadsPerBlock;
    scaleHeatmapKernel<<<blocks, threadsPerBlock>>>(d_heatmap, d_scaled, SIZE, CELLSIZE);
    hipDeviceSynchronize();

    // Allocate a contiguous host array for the scaled heatmap.
    int *shm_flat = (int*)malloc(numScaled * sizeof(int));
    safe_call(hipMemcpy(shm_flat, d_scaled, numScaled * sizeof(int), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();

    // Reassign the 'scaled_heatmap' pointer-to-pointers using the flat array.
    // (Note: You must ensure that later code uses the same layout.)
    for (int i = 0; i < SCALED_SIZE; i++) {
        scaled_heatmap[i] = shm_flat + i * SCALED_SIZE;
    }

    // Free temporary memory.
    free(hm_flat);
    // Do not free shm_flat here because scaled_heatmap[i] pointers refer into it.
    safe_call(hipFree(d_heatmap));
    safe_call(hipFree(d_scaled));
}


// Updates the heatmap according to the agent positions
// void Ped::Model::updateHeatmap()
// {
// 	for (int x = 0; x < SIZE; x++)
// 	{
// 		for (int y = 0; y < SIZE; y++)
// 		{
// 			// heat fades
// 			heatmap[y][x] = (int)round(heatmap[y][x] * 0.80);
// 		}
// 	}

// 	// Count how many agents want to go to each location
// 	for (int i = 0; i < agents->x.size(); i++)
// 	{
// 		//Ped::Tagent* agent = agents->desiredX[i];
// 		int x = agents->desiredX[i];
// 		int y = agents->desiredY[i];

// 		if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
// 		{
// 			continue;
// 		}

// 		// intensify heat for better color results
// 		heatmap[y][x] += 40;

// 	}

// 	for (int x = 0; x < SIZE; x++)
// 	{
// 		for (int y = 0; y < SIZE; y++)
// 		{
// 			heatmap[y][x] = heatmap[y][x] < 255 ? heatmap[y][x] : 255;
// 		}
// 	}

// 	// Scale the data for visual representation
// 	// for (int y = 0; y < SIZE; y++)
// 	// {
// 	// 	for (int x = 0; x < SIZE; x++)
// 	// 	{
// 	// 		int value = heatmap[y][x];
// 	// 		for (int cellY = 0; cellY < CELLSIZE; cellY++)
// 	// 		{
// 	// 			for (int cellX = 0; cellX < CELLSIZE; cellX++)
// 	// 			{
// 	// 				scaled_heatmap[y * CELLSIZE + cellY][x * CELLSIZE + cellX] = value;
// 	// 			}
// 	// 		}
// 	// 	}
// 	// }
//     scaleHeatmapCUDA();
// }

void Ped::Model::updateHeatmap() {
    for (int x = 0; x < SIZE; x++) {
        for (int y = 0; y < SIZE; y++) {
            // heat fades
            heatmap[y][x] = (int)round(heatmap[y][x] * 0.80);
        }
    }

    // Count how many agents want to go to each location
    for (int i = 0; i < agents->x.size(); i++) {
        int x = agents->desiredX[i];
        int y = agents->desiredY[i];

        if (x < 0 || x >= SIZE || y < 0 || y >= SIZE) {
            continue;
        }

        // Intensify heat for better color results
        heatmap[y][x] += 40;
    }

    for (int x = 0; x < SIZE; x++) {
        for (int y = 0; y < SIZE; y++) {
            heatmap[y][x] = heatmap[y][x] < 255 ? heatmap[y][x] : 255;
        }
    }

    // Scale the heatmap using existing CUDA scaling
    scaleHeatmapCUDA();

    // Apply the parallelized CUDA blur filter
    applyBlurFilterCUDA();
}




void Ped::Model::applyBlurFilterCUDA() {
    int numElements = SCALED_SIZE * SCALED_SIZE;

    int *d_scaled_heatmap, *d_blurred_heatmap, *d_weights;
    int weights[5][5] = {
        {1, 4, 7, 4, 1},
        {4, 16, 26, 16, 4},
        {7, 26, 41, 26, 7},
        {4, 16, 26, 16, 4},
        {1, 4, 7, 4, 1}
    };

    // Flatten the weights
    int h_weights[25];
    for (int i = 0; i < 5; i++)
        for (int j = 0; j < 5; j++)
            h_weights[i * 5 + j] = weights[i][j];

    safe_call(hipMalloc(&d_scaled_heatmap, numElements * sizeof(int)));
    safe_call(hipMalloc(&d_blurred_heatmap, numElements * sizeof(int)));
    safe_call(hipMalloc(&d_weights, 25 * sizeof(int)));

    // Flatten the heatmap
    int *h_scaled_heatmap = new int[numElements];
    for (int i = 0; i < SCALED_SIZE; i++)
        memcpy(h_scaled_heatmap + i * SCALED_SIZE, scaled_heatmap[i], SCALED_SIZE * sizeof(int));

    hipMemcpy(d_scaled_heatmap, h_scaled_heatmap, numElements * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, h_weights, 25 * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((SCALED_SIZE + blockDim.x - 1) / blockDim.x, (SCALED_SIZE + blockDim.y - 1) / blockDim.y);

    blurHeatmapKernel<<<gridDim, blockDim>>>(d_scaled_heatmap, d_blurred_heatmap, SCALED_SIZE, d_weights);
    hipDeviceSynchronize();

    // Copy the result back to host
    int *h_blurred_heatmap = new int[numElements];
    hipMemcpy(h_blurred_heatmap, d_blurred_heatmap, numElements * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for (int i = 0; i < SCALED_SIZE; i++) {
        memcpy(blurred_heatmap[i], h_blurred_heatmap + i * SCALED_SIZE, SCALED_SIZE * sizeof(int));
    }

    delete[] h_scaled_heatmap;
    delete[] h_blurred_heatmap;

    hipFree(d_scaled_heatmap);
    hipFree(d_blurred_heatmap);
    hipFree(d_weights);
}

	// Weights for blur filter
	// const int w[5][5] = {
	// 	{ 1, 4, 7, 4, 1 },
	// 	{ 4, 16, 26, 16, 4 },
	// 	{ 7, 26, 41, 26, 7 },
	// 	{ 4, 16, 26, 16, 4 },
	// 	{ 1, 4, 7, 4, 1 }
	// };

// #define WEIGHTSUM 273
// 	// Apply gaussian blurfilter		       
// 	for (int i = 2; i < SCALED_SIZE - 2; i++)
// 	{
// 		for (int j = 2; j < SCALED_SIZE - 2; j++)
// 		{
// 			int sum = 0;
// 			for (int k = -2; k < 3; k++)
// 			{
// 				for (int l = -2; l < 3; l++)
// 				{
// 					sum += w[2 + k][2 + l] * scaled_heatmap[i + k][j + l];
// 				}
// 			}
// 			int value = sum / WEIGHTSUM;
// 			blurred_heatmap[i][j] = 0x00FF0000 | value << 24;
// 		}
// 	}
// }

// int Ped::Model::getHeatmapSize() const {
// 	return SCALED_SIZE;
// }
